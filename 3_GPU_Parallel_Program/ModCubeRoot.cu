
#include <hip/hip_runtime.h>
//******************************************************************************
//
// File:    ModCubRoot.cu
//
// Version: 1.0
//******************************************************************************


// Number of threads per block.
#define NT 1024

// Overall counter variable in global memory.
__device__ unsigned long long int count;
__device__ unsigned long long int arraySize = 3;


/**
 * Device kernel to compute modular cube root.
 *
 *
 * @author  Omkar Kakade
 */
extern "C" __global__ void computeModularCubeRoot
   ( unsigned long long int c,
     unsigned long long int N,
     unsigned long long int *final_M)
   {
   unsigned long long int thr, size, rank;
   unsigned long long int local_c;
   unsigned long long int local_m;
   unsigned long long int increment;
   // Determine number of threads and this thread's rank.
   thr = threadIdx.x;
   size = gridDim.x*NT;
   rank = blockIdx.x*NT + thr;
   
   // Initialize per-thread.
   local_c = 0;
   local_m = 0;
   increment = 1;
   
   unsigned long long int atom_result =0;

   // Compute modular cube roots.
   for (unsigned long long int i = rank; i < N; i += size)
      {
      unsigned long long int first_mod = (i)%N;
      unsigned long long int second_mod = (first_mod * i)%N;
      unsigned long long int third_mod = (second_mod * i)%N;
      local_c = third_mod;
      local_m = i;
      
    
      if (local_c == c){
	  
	  // atomic counter value updation.
	  atom_result = atomicAdd(&count,increment);
	  
	  if (atom_result < arraySize) {
	  	
          	final_M[atom_result]=local_m;
		
    	}
  
	}
      }
 

}
  
   
